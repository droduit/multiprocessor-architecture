/*
 ============================================================================
 Filename    : assignment4.c
 Author      : Arash Pourhabibi, Hussein Kassir
 ============================================================================
 */

#include <iostream>
#include <iomanip>
#include <fstream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;
#include "utility.h"

void array_process(double *input, double *output, int length, int iterations);
void GPU_array_process(double *input, double *output, int length, int iterations);

int main (int argc, const char *argv[]) {

    int length, iterations;
    double time;

    if (argc != 3) {
		cout<<"Invalid input!"<<endl<<"Usage: ./assignment4 <length> <iterations>"<<endl;
		return 1;
	} else {
        length      = atoi(argv[1]);
        iterations  = atoi(argv[2]);
        if(length%2!=0)
        {
            cout<<"Invalid input!"<<endl<<"Array length must be even"<<endl;
            return 1;
        }
	}


    //Allocate arrays
    double *input   = new double[length*length];
    double *output  = new double[length*length];

    //Reset Device
    hipDeviceReset();

    //Initialize the arrays
    init(input, length);
    init(output, length);

    //Start timer
    set_clock();

    /*Use either the CPU or the GPU functions*/

    //CPU Baseline
    //Uncomment the block to use the baseline
    array_process(input, output, length, iterations);
    if(iterations%2==0)
    {
        double *temp;
        temp = input;
        input = output;
        output = temp;
    }

    //GPU function
    //GPU_array_process(input, output,  length, iterations);

    //Stop timer
    time = elapsed_time();

    //Report time required for n iterations
    cout<<"Running the algorithm on "<<length<<" by "<<length<<" array for "<<iterations<<" iteration takes "<<setprecision(4)<<time<<"s"<<endl;

    //Save array in filelength
    save(output, length);

    //Free allocated memory
    delete[] input;
    delete[] output;

    return 0;
}
